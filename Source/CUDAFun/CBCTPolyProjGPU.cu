#include "hip/hip_runtime.h"


#include "CBCTPolyProjGPU.cuh"

#include <iostream>


hipTextureObject_t texObj = 0;
hipArray_t d_cuArray3D;
hipExtent volumeSize;

void forwardProjGridGPU(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	// Select Gpu
	/*hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}*/

	//initDeviceVar(d_mPolyForwardProj, d_mCoordinate, mCTScanSystemInfo, mCTScanParas, h_mPolyForwardProj);

	//hipTextureObject_t texObj = 0;
	// ģ���������
	updateTex(d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);

	// դ���� ���� -> �Դ�
	hipMemcpy(d_mPolyForwardProj.grid, h_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float), hipMemcpyHostToDevice);
	// ̽������Ӧ���� ���� -> �Դ�
	hipMemcpy(d_mPolyForwardProj.detResponse, h_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyHostToDevice);


	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);


	// Kernel parameters
	dim3 blockSizeTrans(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeTrans((mCTScanSystemInfo.intNum - 1) / blockSizeTrans.x + 1, (mCTScanParas.dNumU - 1) / blockSizeTrans.y + 1, (mCTScanParas.dNumV - 1) / blockSizeTrans.z + 1);

	dim3 blockSizeProj(BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1);

	float angle = 0.0f;
	for (size_t i = 0; i < mCTScanParas.projNum; i++)
	{

		angle = mCTScanSystemInfo.rotatedDirection * i * mCTScanSystemInfo.thetaStep / 180 * PI;   // ���㵱ǰ��ת�ǣ���ת��Ϊ������

		transformKernel << <gridSizeTrans, blockSizeTrans >> > (d_mPolyForwardProj, d_mCoordinate, texObj, mCTScanSystemInfo, mCTScanParas, angle);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "transformKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transformKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
		//hipMemcpy(tempIntX, d_mPolyForwardProj->phantom, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);
		//FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(tempIntX, 1, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), fp);
		//fclose(fp);

		forwardProjGridKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas, i);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "forwardProjKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardProjKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//hipMemcpy(h_mPolyForwardProj->I, d_mPolyForwardProj->I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
		//// ���
		////FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(h_mPolyForwardProj->I, 1, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), fp);
		//fclose(fp);
	}

	// I  �Դ� -> ����
	hipMemcpy(h_mPolyForwardProj.I, d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_mPolyForwardProj.IAbsorb, d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);


	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);

	//// ���Դ���
	//FILE* fp;
	//fp = fopen("test.raw", "wb");
	//fwrite(h_mPolyForwardProj->I, 1, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), fp);
	//fclose(fp);
}


//// ��դ��������ͶӰ
//void forwardSinMatProjGridGPU(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
//{
//	//// ģ���������
//	//updateTex(d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);  // �����ܶ�
//
//	// դ���� ���� -> �Դ�
//	hipMemcpy(d_mPolyForwardProj.grid, h_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float), hipMemcpyHostToDevice);
//	
//	// ̽������Ӧ���� ���� -> �Դ�
//	hipMemcpy(d_mPolyForwardProj.detResponse, h_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyHostToDevice);
//
//
//	// GPU��ʱ
//	hipEvent_t g_start, g_stop;
//	hipEventCreate(&g_start);
//	hipEventCreate(&g_stop);
//	hipEventRecord(g_start, 0);
//
//	// Kernel parameters
//	dim3 blockSizeTrans(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
//	dim3 gridSizeTrans((mCTScanSystemInfo.intNum - 1) / blockSizeTrans.x + 1, (mCTScanParas.dNumU - 1) / blockSizeTrans.y + 1, (mCTScanParas.dNumV - 1) / blockSizeTrans.z + 1);
//
//	dim3 blockSizeProj(BLOCKSIZEY, BLOCKSIZEZ);
//	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1);
//
//	float angle = 0.0f;
//	for (size_t i = 0; i < mCTScanParas.projNum; i++)
//	{
//
//		angle = mCTScanSystemInfo.rotatedDirection * i * mCTScanSystemInfo.thetaStep / 180 * PI;   // ���㵱ǰ��ת�ǣ���ת��Ϊ������
//
//		transformKernel << <gridSizeTrans, blockSizeTrans >> > (d_mPolyForwardProj, d_mCoordinate, texObj, mCTScanSystemInfo, mCTScanParas, angle);
//
//		hipError_t cudaStatus = hipGetLastError();
//		if (cudaStatus != hipSuccess) {
//			fprintf(stderr, "transformKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//		}cudaStatus = hipDeviceSynchronize();
//		if (cudaStatus != hipSuccess) {
//			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transformKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
//		}
//
//		//// ���Դ���
//		//float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
//		//hipMemcpy(tempIntX, d_mPolyForwardProj->phantom, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);
//		//FILE* fp;
//		//fp = fopen("test.raw", "wb");
//		//fwrite(tempIntX, 1, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), fp);
//		//fclose(fp);
//
//		forwardSinMatProjGridKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas, i);
//
//		cudaStatus = hipGetLastError();
//		if (cudaStatus != hipSuccess) {
//			fprintf(stderr, "forwardProjKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//		}cudaStatus = hipDeviceSynchronize();
//		if (cudaStatus != hipSuccess) {
//			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardProjKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
//		}
//
//		//// ���Դ���
//		//hipMemcpy(h_mPolyForwardProj->I, d_mPolyForwardProj->I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
//		//// ���
//		////FILE* fp;
//		//fp = fopen("test.raw", "wb");
//		//fwrite(h_mPolyForwardProj->I, 1, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), fp);
//		//fclose(fp);
//	}
//
//	// I  �Դ� -> ����
//	hipMemcpy(h_mPolyForwardProj.I, d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
//	hipMemcpy(h_mPolyForwardProj.IAbsorb, d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
//
//
//	// ��ʱ
//	hipEventRecord(g_stop, 0);
//	hipEventSynchronize(g_stop);
//	float elapsedTime = 0;
//	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
//	std::cout << "==>>��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
//	hipEventDestroy(g_start);
//	hipEventDestroy(g_stop);
//
//}

// ��դ��������ͶӰ
void forwardSinMatProjGridGPU(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	//// ģ���������
	//updateTex(d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);  // �����ܶ�

	// դ���� ���� -> �Դ�
	hipMemcpy(d_mPolyForwardProj.grid, h_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float), hipMemcpyHostToDevice);

	// ̽������Ӧ���� ���� -> �Դ�
	hipMemcpy(d_mPolyForwardProj.detResponse, h_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyHostToDevice);


	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	// Kernel parameters
	dim3 blockSizeProj(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1, (mCTScanParas.projNum - 1) / blockSizeProj.y + 1);

	forwardSinMatProjGridKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "forwardSinMatProjGridKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardSinMatProjGridKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// I  �Դ� -> ����
	hipMemcpy(h_mPolyForwardProj.I, d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_mPolyForwardProj.IAbsorb, d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);


	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);

}

void forwardSinMatNoResponseProjGridGPU(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	// դ���� ���� -> �Դ�
	hipMemcpy(d_mPolyForwardProj.grid, h_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float), hipMemcpyHostToDevice);

	// ̽������Ӧ���� ���� -> �Դ�
	// hipMemcpy(d_mPolyForwardProj.detResponse, h_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyHostToDevice);


	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	// Kernel parameters
	dim3 blockSizeProj(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1, (mCTScanParas.projNum - 1) / blockSizeProj.y + 1);

	forwardSinMatNoResponseProjGridKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "forwardSinMatNoResponseProjGridKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardSinMatNoResponseProjGridKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// I  �Դ� -> ����
	hipMemcpy(h_mPolyForwardProj.I, d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_mPolyForwardProj.IAbsorb, d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);


	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);
}


// ��դ��ͶӰ
void forwardProjNoGridGPU(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	//// Select Gpu
	//hipError_t cudaStatus;
	//cudaStatus = hipSetDevice(0);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	//}

	//initDeviceVar(d_mPolyForwardProj, d_mCoordinate, mCTScanSystemInfo, mCTScanParas, h_mPolyForwardProj);

	//hipTextureObject_t texObj = 0;
	// ģ���������
	updateTex(d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);

	// ̽������Ӧ���� ���� -> �Դ�
	hipMemcpy(d_mPolyForwardProj.detResponse, h_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyHostToDevice);


	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);


	// Kernel parameters
	dim3 blockSizeTrans(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeTrans((mCTScanSystemInfo.intNum - 1) / blockSizeTrans.x + 1, (mCTScanParas.dNumU - 1) / blockSizeTrans.y + 1, (mCTScanParas.dNumV - 1) / blockSizeTrans.z + 1);

	dim3 blockSizeProj(BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1);

	float angle = 0.0f;
	for (size_t i = 0; i < mCTScanParas.projNum; i++)
	{

		angle = mCTScanSystemInfo.rotatedDirection * i * mCTScanSystemInfo.thetaStep / 180 * PI;   // ���㵱ǰ��ת�ǣ���ת��Ϊ������

		transformKernel << <gridSizeTrans, blockSizeTrans >> > (d_mPolyForwardProj, d_mCoordinate, texObj, mCTScanSystemInfo, mCTScanParas, angle);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "transformKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transformKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
		//hipMemcpy(tempIntX, d_mPolyForwardProj->phantom, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);
		//FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(tempIntX, 1, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), fp);
		//fclose(fp);

		forwardProjNoGridKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas, i);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "forwardProjKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardProjKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//hipMemcpy(h_mPolyForwardProj->I, d_mPolyForwardProj->I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
		//// ���
		////FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(h_mPolyForwardProj->I, 1, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), fp);
		//fclose(fp);
	}

	// I  �Դ� -> ����
	hipMemcpy(h_mPolyForwardProj.I, d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_mPolyForwardProj.IAbsorb, d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);


	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);
}


// ��դ��������ͶӰ
void forwardSinMatProjNoGridGPU(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{


	//// ģ���������
	//updateTex(d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);

	// ̽������Ӧ���� ���� -> �Դ�
	hipMemcpy(d_mPolyForwardProj.detResponse, h_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyHostToDevice);


	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	// Kernel parameters
	dim3 blockSizeProj(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1, (mCTScanParas.projNum - 1) / blockSizeProj.y + 1);

	forwardSinMatProjNoGridKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "forwardSinMatProjNoGridKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardSinMatProjNoGridKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	// I  �Դ� -> ����
	hipMemcpy(h_mPolyForwardProj.I, d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_mPolyForwardProj.IAbsorb, d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);


	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);
}

void forwardSinMatNoResponseProjNoGridGPU(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	// դ���� ���� -> �Դ�
	//hipMemcpy(d_mPolyForwardProj.grid, h_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float), hipMemcpyHostToDevice);

	// ̽������Ӧ���� ���� -> �Դ�
	// hipMemcpy(d_mPolyForwardProj.detResponse, h_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyHostToDevice);


	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	// Kernel parameters
	dim3 blockSizeProj(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1, (mCTScanParas.projNum - 1) / blockSizeProj.y + 1);

	forwardSinMatNoResponseProjNoGridKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas);

	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "forwardSinMatNoResponseProjNoGridKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardSinMatNoResponseProjNoGridKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// I  �Դ� -> ����
	hipMemcpy(h_mPolyForwardProj.I, d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_mPolyForwardProj.IAbsorb, d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);


	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);
}



void initDeviceGrid(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	hipError_t cudaStatus;

	// Select Gpu
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed? \n Error: %s\n", hipGetErrorString(cudaStatus));
	}

	// ��ʼ��device
	hipDeviceSynchronize();    // ȷ����CUDA��������
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(cudaStatus));
	}
	hipDeviceReset();  // ������


	// Allocate GPU buffers
	size_t sizeIntPoint = mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float);

	// ���ֵ�����
	cudaStatus = hipMalloc(&d_mCoordinate.imgIntX, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->x hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntY, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->y hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntZ, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->z hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	/*hipMalloc(&d_mCoordinate->detU, mCTScanParas.dNumU * sizeof(float));
	hipMalloc(&d_mCoordinate->detV, mCTScanParas.dNumV * sizeof(float));*/

	//hipMallocManaged();

	dim3 blockSizeIPC(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeIPC((mCTScanSystemInfo.intNum - 1) / blockSizeIPC.x + 10, (mCTScanParas.dNumU - 1) / blockSizeIPC.y + 10, (mCTScanParas.dNumV - 1) / blockSizeIPC.z + 10);

	computeIntPointCoordinatesKernel << <gridSizeIPC, blockSizeIPC >> > (d_mCoordinate, mCTScanSystemInfo, mCTScanParas);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeIntPointCoordinatesKernel launch failed: %s! \n\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeIntPointCoordinatesKernel!\n Error: %s!\n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	// ���
	/*float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntX, d_mCoordinate->imgIntX, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntY = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntY, d_mCoordinate->imgIntY, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntZ = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntZ, d_mCoordinate->imgIntZ, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);*/


	// ��ʼ������
	volumeSize = make_hipExtent(mCTScanParas.pNumX, mCTScanParas.pNumY, mCTScanParas.pNumZ);
	initTexture3D(texObj, d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);


	// Allocate GPU buffers for temporary output variables
	cudaStatus = hipMalloc(&d_mPolyForwardProj.phantom, sizeIntPoint);   // �洢���ֵ㴦����ֵ��ת�������
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.phantom hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}
	// Allocate GPU buffers and host
	hipMalloc(&d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));
	hipMalloc(&d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));

	// դ����
	hipMalloc(&d_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float));

	// ̽������Ӧ����
	hipMalloc(&d_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float));
}

void initDeviceSinMatGrid(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	hipError_t cudaStatus;

	// Select Gpu
	cudaStatus = hipSetDevice(GPUINDEX);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed? \n Error: %s\n", hipGetErrorString(cudaStatus));
	}

	// ��ʼ��device
	hipDeviceSynchronize();    // ȷ����CUDA��������
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(cudaStatus));
	}
	hipDeviceReset();  // ������


	// Allocate GPU buffers
	size_t sizeIntPoint = mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float);

	// ���ֵ�����
	cudaStatus = hipMalloc(&d_mCoordinate.imgIntX, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->x hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntY, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->y hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntZ, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->z hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	/*hipMalloc(&d_mCoordinate->detU, mCTScanParas.dNumU * sizeof(float));
	hipMalloc(&d_mCoordinate->detV, mCTScanParas.dNumV * sizeof(float));*/

	//hipMallocManaged();

	dim3 blockSizeIPC(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeIPC((mCTScanSystemInfo.intNum - 1) / blockSizeIPC.x + 10, (mCTScanParas.dNumU - 1) / blockSizeIPC.y + 10, (mCTScanParas.dNumV - 1) / blockSizeIPC.z + 10);

	computeIntPointCoordinatesKernel << <gridSizeIPC, blockSizeIPC >> > (d_mCoordinate, mCTScanSystemInfo, mCTScanParas);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeIntPointCoordinatesKernel launch failed: %s! \n\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeIntPointCoordinatesKernel!\n Error: %s!\n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���
	/*float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntX, d_mCoordinate->imgIntX, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntY = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntY, d_mCoordinate->imgIntY, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntZ = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntZ, d_mCoordinate->imgIntZ, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);*/


	// ��ʼ������
	volumeSize = make_hipExtent(mCTScanParas.pNumX, mCTScanParas.pNumY, mCTScanParas.pNumZ);
	initTexture3D(texObj, d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);


	// Allocate GPU buffers for temporary output variables
	cudaStatus = hipMalloc(&d_mPolyForwardProj.phantom, sizeIntPoint);   // �洢���ֵ㴦����ֵ��ת�������
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.phantom hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���� �ܶ�ͼ���Proj
	// �ܶ�ͼ���Proj
	hipMalloc(&d_mPolyForwardProj.proj, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.proj hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	// Kernel parameters
	dim3 blockSizeTrans(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeTrans((mCTScanSystemInfo.intNum - 1) / blockSizeTrans.x + 1, (mCTScanParas.dNumU - 1) / blockSizeTrans.y + 1, (mCTScanParas.dNumV - 1) / blockSizeTrans.z + 1);

	dim3 blockSizeProj(BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1);

	float angle = 0.0f;
	for (size_t i = 0; i < mCTScanParas.projNum; i++)
	{

		angle = mCTScanSystemInfo.rotatedDirection * i * mCTScanSystemInfo.thetaStep / 180 * PI;   // ���㵱ǰ��ת�ǣ���ת��Ϊ������

		transformKernel << <gridSizeTrans, blockSizeTrans >> > (d_mPolyForwardProj, d_mCoordinate, texObj, mCTScanSystemInfo, mCTScanParas, angle);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "transformKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transformKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
		//hipMemcpy(tempIntX, d_mPolyForwardProj->phantom, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);
		//FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(tempIntX, 1, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), fp);
		//fclose(fp);

		computeSinMatIndensityProjKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas, i);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "forwardProjKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardProjKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}


	}

	// ���Դ���
	//h_mPolyForwardProj.proj = new float[mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum];
	//hipMemcpy(h_mPolyForwardProj.proj, d_mPolyForwardProj.proj, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
	//// ���
	//FILE* fp;
	//fp = fopen("test.raw", "wb");
	//fwrite(h_mPolyForwardProj.proj, 1, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), fp);
	//fclose(fp);

	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>�ܶ�ͼ��ͶӰ��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);



	// Allocate GPU buffers and host
	hipMalloc(&d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));
	hipMalloc(&d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));

	// դ����
	hipMalloc(&d_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float));

	// ̽������Ӧ����
	hipMalloc(&d_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float));
}

// ��ʼ���������ʡ���դ������Ӧ���н���
void initDeviceSinMatFoSpSiGrid(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	hipError_t cudaStatus;

	// Select Gpu
	cudaStatus = hipSetDevice(GPUINDEX);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed? \n Error: %s\n", hipGetErrorString(cudaStatus));
	}

	// ��ʼ��device
	hipDeviceSynchronize();    // ȷ����CUDA��������
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(cudaStatus));
	}
	hipDeviceReset();  // ������


	// Allocate GPU buffers
	size_t sizeIntPoint = mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float);

	// ���ֵ�����
	cudaStatus = hipMalloc(&d_mCoordinate.imgIntX, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->x hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntY, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->y hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntZ, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->z hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	/*hipMalloc(&d_mCoordinate->detU, mCTScanParas.dNumU * sizeof(float));
	hipMalloc(&d_mCoordinate->detV, mCTScanParas.dNumV * sizeof(float));*/

	//hipMallocManaged();

	// ������ֵ�����
	dim3 blockSizeIPC(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeIPC((mCTScanSystemInfo.intNum - 1) / blockSizeIPC.x + 10, (mCTScanParas.dNumU - 1) / blockSizeIPC.y + 10, (mCTScanParas.dNumV - 1) / blockSizeIPC.z + 10);

	computeIntPointCoordinatesKernel << <gridSizeIPC, blockSizeIPC >> > (d_mCoordinate, mCTScanSystemInfo, mCTScanParas);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeIntPointCoordinatesKernel launch failed: %s! \n\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeIntPointCoordinatesKernel!\n Error: %s!\n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���
	/*float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntX, d_mCoordinate->imgIntX, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntY = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntY, d_mCoordinate->imgIntY, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntZ = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntZ, d_mCoordinate->imgIntZ, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);*/


	// ��ʼ������
	volumeSize = make_hipExtent(mCTScanParas.pNumX, mCTScanParas.pNumY, mCTScanParas.pNumZ);
	initTexture3D(texObj, d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);


	// Allocate GPU buffers for temporary output variables
	cudaStatus = hipMalloc(&d_mPolyForwardProj.phantom, sizeIntPoint);   // �洢���ֵ㴦����ֵ��ת�������
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.phantom hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���� �ܶ�ͼ���Proj
	// �ܶ�ͼ���Proj
	cudaStatus = hipMalloc(&d_mPolyForwardProj.proj, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.proj hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ����ƫ�����ݷ���
	size_t sizeFoSpOffset = mCTScanParas.projNum * sizeof(float);

	cudaStatus = hipMalloc(&d_mPolyForwardProj.foSpOffsetU, sizeFoSpOffset);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.foSpOffsetU hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mPolyForwardProj.foSpOffsetV, sizeFoSpOffset);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.foSpOffsetV hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ��������
	cudaStatus = hipMemcpy(d_mPolyForwardProj.foSpOffsetU, h_mPolyForwardProj.foSpOffsetU, sizeFoSpOffset, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.foSpOffsetU hipMemcpy failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMemcpy(d_mPolyForwardProj.foSpOffsetV, h_mPolyForwardProj.foSpOffsetV, sizeFoSpOffset, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.foSpOffsetV hipMemcpy failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	// Kernel parameters
	dim3 blockSizeTrans(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeTrans((mCTScanSystemInfo.intNum - 1) / blockSizeTrans.x + 1, (mCTScanParas.dNumU - 1) / blockSizeTrans.y + 1, (mCTScanParas.dNumV - 1) / blockSizeTrans.z + 1);

	dim3 blockSizeProj(BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1);

	float angle = 0.0f;
	for (size_t i = 0; i < mCTScanParas.projNum; i++)
	{

		//angle = mCTScanSystemInfo.rotatedDirection * i * mCTScanSystemInfo.thetaStep / 180 * PI;   // ���㵱ǰ��ת�ǣ���ת��Ϊ������

		transformFocalSpotKernel << <gridSizeTrans, blockSizeTrans >> > (d_mPolyForwardProj, d_mCoordinate, texObj, mCTScanSystemInfo, mCTScanParas, i);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "transformFocalSpotKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transformFocalSpotKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
		//hipMemcpy(tempIntX, d_mPolyForwardProj->phantom, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);
		//FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(tempIntX, 1, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), fp);
		//fclose(fp);

		computeSinMatIndensityProjKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas, i);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "forwardProjKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardProjKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//hipMemcpy(h_mPolyForwardProj->I, d_mPolyForwardProj->I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
		//// ���
		////FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(h_mPolyForwardProj->I, 1, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), fp);
		//fclose(fp);
	}

	// ����ͶӰ����ƫ�ƣ���������ƫ�ƣ��൱�������̽����ͬʱ��ͬһ����ƫ��
	createTexture3D(texObj, d_mPolyForwardProj.proj, mCTScanParas.dNumU, mCTScanParas.dNumV, mCTScanParas.projNum);

	dim3 blockSizePOffsetMatch(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizePOffsetMatch((mCTScanParas.dNumU - 1) / blockSizePOffsetMatch.x + 1, (mCTScanParas.dNumV - 1) / blockSizePOffsetMatch.y + 1, (mCTScanParas.projNum - 1) / blockSizePOffsetMatch.z + 1);

	projOffsetMatchKernel << <gridSizePOffsetMatch, blockSizePOffsetMatch >> > (d_mPolyForwardProj, texObj, mCTScanParas);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "projOffsetMatchKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching projOffsetMatchKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>�ܶ�ͼ��ͶӰ��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);



	// Allocate GPU buffers and host
	hipMalloc(&d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));
	hipMalloc(&d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));

	// դ����
	hipMalloc(&d_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float));

	// ̽������Ӧ����
	hipMalloc(&d_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float));

}

void initDeviceSinMatNoResponseGrid(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	hipError_t cudaStatus;

	// Select Gpu
	cudaStatus = hipSetDevice(GPUINDEX);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed? \n Error: %s\n", hipGetErrorString(cudaStatus));
	}

	// ��ʼ��device
	hipDeviceSynchronize();    // ȷ����CUDA��������
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(cudaStatus));
	}
	hipDeviceReset();  // ������


	// Allocate GPU buffers
	size_t sizeIntPoint = mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float);

	// ���ֵ�����
	cudaStatus = hipMalloc(&d_mCoordinate.imgIntX, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->x hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntY, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->y hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntZ, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->z hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	/*hipMalloc(&d_mCoordinate->detU, mCTScanParas.dNumU * sizeof(float));
	hipMalloc(&d_mCoordinate->detV, mCTScanParas.dNumV * sizeof(float));*/

	//hipMallocManaged();

	dim3 blockSizeIPC(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeIPC((mCTScanSystemInfo.intNum - 1) / blockSizeIPC.x + 10, (mCTScanParas.dNumU - 1) / blockSizeIPC.y + 10, (mCTScanParas.dNumV - 1) / blockSizeIPC.z + 10);

	computeIntPointCoordinatesKernel << <gridSizeIPC, blockSizeIPC >> > (d_mCoordinate, mCTScanSystemInfo, mCTScanParas);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeIntPointCoordinatesKernel launch failed: %s! \n\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeIntPointCoordinatesKernel!\n Error: %s!\n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���
	/*float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntX, d_mCoordinate->imgIntX, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntY = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntY, d_mCoordinate->imgIntY, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntZ = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntZ, d_mCoordinate->imgIntZ, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);*/


	// ��ʼ������
	volumeSize = make_hipExtent(mCTScanParas.pNumX, mCTScanParas.pNumY, mCTScanParas.pNumZ);
	initTexture3D(texObj, d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);


	// Allocate GPU buffers for temporary output variables
	cudaStatus = hipMalloc(&d_mPolyForwardProj.phantom, sizeIntPoint);   // �洢���ֵ㴦����ֵ��ת�������
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.phantom hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���� �ܶ�ͼ���Proj
	// �ܶ�ͼ���Proj
	hipMalloc(&d_mPolyForwardProj.proj, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.proj hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	// Kernel parameters
	dim3 blockSizeTrans(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeTrans((mCTScanSystemInfo.intNum - 1) / blockSizeTrans.x + 1, (mCTScanParas.dNumU - 1) / blockSizeTrans.y + 1, (mCTScanParas.dNumV - 1) / blockSizeTrans.z + 1);

	dim3 blockSizeProj(BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1);

	float angle = 0.0f;
	for (size_t i = 0; i < mCTScanParas.projNum; i++)
	{

		angle = mCTScanSystemInfo.rotatedDirection * i * mCTScanSystemInfo.thetaStep / 180 * PI;   // ���㵱ǰ��ת�ǣ���ת��Ϊ������

		transformKernel << <gridSizeTrans, blockSizeTrans >> > (d_mPolyForwardProj, d_mCoordinate, texObj, mCTScanSystemInfo, mCTScanParas, angle);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "transformKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transformKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
		//hipMemcpy(tempIntX, d_mPolyForwardProj->phantom, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);
		//FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(tempIntX, 1, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), fp);
		//fclose(fp);

		computeSinMatIndensityProjKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas, i);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "forwardProjKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardProjKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//hipMemcpy(h_mPolyForwardProj->I, d_mPolyForwardProj->I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
		//// ���
		////FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(h_mPolyForwardProj->I, 1, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), fp);
		//fclose(fp);
	}

	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>�ܶ�ͼ��ͶӰ��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);



	// Allocate GPU buffers and host
	hipMalloc(&d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));
	hipMalloc(&d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));

	// դ����
	hipMalloc(&d_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float));

}

void initDeviceNoGrid(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	// ��ʼ��device
	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(cudaStatus));
	}
	hipDeviceReset();

	// Select Gpu
	//hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	// Allocate GPU buffers
	size_t sizeIntPoint = mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float);

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntX, sizeIntPoint);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable imageCoordinate->x hipMalloc failed!");
	}
	hipMalloc(&d_mCoordinate.imgIntY, sizeIntPoint);
	hipMalloc(&d_mCoordinate.imgIntZ, sizeIntPoint);


	/*hipMalloc(&d_mCoordinate->detU, mCTScanParas.dNumU * sizeof(float));
	hipMalloc(&d_mCoordinate->detV, mCTScanParas.dNumV * sizeof(float));*/

	//hipMallocManaged();

	dim3 blockSizeIPC(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeIPC((mCTScanSystemInfo.intNum - 1) / blockSizeIPC.x + 10, (mCTScanParas.dNumU - 1) / blockSizeIPC.y + 10, (mCTScanParas.dNumV - 1) / blockSizeIPC.z + 10);

	computeIntPointCoordinatesKernel << <gridSizeIPC, blockSizeIPC >> > (d_mCoordinate, mCTScanSystemInfo, mCTScanParas);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeIntPointCoordinatesKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeIntPointCoordinatesKernel!\n Error: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	// ���
	/*float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntX, d_mCoordinate->imgIntX, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntY = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntY, d_mCoordinate->imgIntY, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntZ = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntZ, d_mCoordinate->imgIntZ, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);*/


	// ��ʼ������
	volumeSize = make_hipExtent(mCTScanParas.pNumX, mCTScanParas.pNumY, mCTScanParas.pNumZ);
	initTexture3D(texObj, d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);


	// Allocate GPU buffers for temporary output variables
	cudaStatus = hipMalloc(&d_mPolyForwardProj.phantom, sizeIntPoint);   // �洢���ֵ㴦����ֵ��ת�������
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Variable d_mPolyForwardProj->phantom hipMalloc failed!");
	}
	// Allocate GPU buffers and host
	hipMalloc(&d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));
	hipMalloc(&d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));

	// ̽������Ӧ����
	//mallocDetResponse(d_mPolyForwardProj, mCTScanParas);
	hipMalloc(&d_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float));
}


void initDeviceSinMatNoGrid(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	hipError_t cudaStatus;

	// Select Gpu
	cudaStatus = hipSetDevice(GPUINDEX);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed? \n Error: %s\n", hipGetErrorString(cudaStatus));
	}

	// ��ʼ��device
	hipDeviceSynchronize();    // ȷ����CUDA��������
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(cudaStatus));
	}
	hipDeviceReset();  // ������


	// Allocate GPU buffers
	size_t sizeIntPoint = mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float);

	// ���ֵ�����
	cudaStatus = hipMalloc(&d_mCoordinate.imgIntX, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->x hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntY, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->y hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntZ, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->z hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	/*hipMalloc(&d_mCoordinate->detU, mCTScanParas.dNumU * sizeof(float));
	hipMalloc(&d_mCoordinate->detV, mCTScanParas.dNumV * sizeof(float));*/

	//hipMallocManaged();

	dim3 blockSizeIPC(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeIPC((mCTScanSystemInfo.intNum - 1) / blockSizeIPC.x + 10, (mCTScanParas.dNumU - 1) / blockSizeIPC.y + 10, (mCTScanParas.dNumV - 1) / blockSizeIPC.z + 10);

	computeIntPointCoordinatesKernel << <gridSizeIPC, blockSizeIPC >> > (d_mCoordinate, mCTScanSystemInfo, mCTScanParas);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeIntPointCoordinatesKernel launch failed: %s! \n\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeIntPointCoordinatesKernel!\n Error: %s!\n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���
	/*float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntX, d_mCoordinate->imgIntX, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntY = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntY, d_mCoordinate->imgIntY, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntZ = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntZ, d_mCoordinate->imgIntZ, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);*/


	// ��ʼ������
	volumeSize = make_hipExtent(mCTScanParas.pNumX, mCTScanParas.pNumY, mCTScanParas.pNumZ);
	initTexture3D(texObj, d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);


	// Allocate GPU buffers for temporary output variables
	cudaStatus = hipMalloc(&d_mPolyForwardProj.phantom, sizeIntPoint);   // �洢���ֵ㴦����ֵ��ת�������
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.phantom hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���� �ܶ�ͼ���Proj
	// �ܶ�ͼ���Proj
	hipMalloc(&d_mPolyForwardProj.proj, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.proj hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	// Kernel parameters
	dim3 blockSizeTrans(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeTrans((mCTScanSystemInfo.intNum - 1) / blockSizeTrans.x + 1, (mCTScanParas.dNumU - 1) / blockSizeTrans.y + 1, (mCTScanParas.dNumV - 1) / blockSizeTrans.z + 1);

	dim3 blockSizeProj(BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1);

	float angle = 0.0f;
	for (size_t i = 0; i < mCTScanParas.projNum; i++)
	{

		angle = mCTScanSystemInfo.rotatedDirection * i * mCTScanSystemInfo.thetaStep / 180 * PI;   // ���㵱ǰ��ת�ǣ���ת��Ϊ������

		transformKernel << <gridSizeTrans, blockSizeTrans >> > (d_mPolyForwardProj, d_mCoordinate, texObj, mCTScanSystemInfo, mCTScanParas, angle);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "transformKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transformKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
		//hipMemcpy(tempIntX, d_mPolyForwardProj->phantom, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);
		//FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(tempIntX, 1, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), fp);
		//fclose(fp);

		computeSinMatIndensityProjKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas, i);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "forwardProjKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardProjKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//hipMemcpy(h_mPolyForwardProj->I, d_mPolyForwardProj->I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
		//// ���
		////FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(h_mPolyForwardProj->I, 1, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), fp);
		//fclose(fp);
	}

	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>�ܶ�ͼ��ͶӰ��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);



	// Allocate GPU buffers and host
	hipMalloc(&d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));
	hipMalloc(&d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));

	// դ����
	//hipMalloc(&d_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float));

	// ̽������Ӧ����
	hipMalloc(&d_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float));


}

void initDeviceSinMatFoSpSiNoGrid(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	hipError_t cudaStatus;

	// Select Gpu
	cudaStatus = hipSetDevice(GPUINDEX);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed? \n Error: %s\n", hipGetErrorString(cudaStatus));
	}

	// ��ʼ��device
	hipDeviceSynchronize();    // ȷ����CUDA��������
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(cudaStatus));
	}
	hipDeviceReset();  // ������


	// Allocate GPU buffers
	size_t sizeIntPoint = mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float);

	// ���ֵ�����
	cudaStatus = hipMalloc(&d_mCoordinate.imgIntX, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->x hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntY, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->y hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntZ, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->z hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	/*hipMalloc(&d_mCoordinate->detU, mCTScanParas.dNumU * sizeof(float));
	hipMalloc(&d_mCoordinate->detV, mCTScanParas.dNumV * sizeof(float));*/

	//hipMallocManaged();

	dim3 blockSizeIPC(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeIPC((mCTScanSystemInfo.intNum - 1) / blockSizeIPC.x + 10, (mCTScanParas.dNumU - 1) / blockSizeIPC.y + 10, (mCTScanParas.dNumV - 1) / blockSizeIPC.z + 10);

	computeIntPointCoordinatesKernel << <gridSizeIPC, blockSizeIPC >> > (d_mCoordinate, mCTScanSystemInfo, mCTScanParas);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeIntPointCoordinatesKernel launch failed: %s! \n\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeIntPointCoordinatesKernel!\n Error: %s!\n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���
	/*float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntX, d_mCoordinate->imgIntX, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntY = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntY, d_mCoordinate->imgIntY, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntZ = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntZ, d_mCoordinate->imgIntZ, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);*/


	// ��ʼ������
	volumeSize = make_hipExtent(mCTScanParas.pNumX, mCTScanParas.pNumY, mCTScanParas.pNumZ);
	initTexture3D(texObj, d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);


	// Allocate GPU buffers for temporary output variables
	cudaStatus = hipMalloc(&d_mPolyForwardProj.phantom, sizeIntPoint);   // �洢���ֵ㴦����ֵ��ת�������
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.phantom hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���� �ܶ�ͼ���Proj
	// �ܶ�ͼ���Proj
	hipMalloc(&d_mPolyForwardProj.proj, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.proj hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ����ƫ�����ݷ���
	size_t sizeFoSpOffset = mCTScanParas.projNum * sizeof(float);

	cudaStatus = hipMalloc(&d_mPolyForwardProj.foSpOffsetU, sizeFoSpOffset);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.foSpOffsetU hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mPolyForwardProj.foSpOffsetV, sizeFoSpOffset);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.foSpOffsetV hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ��������
	cudaStatus = hipMemcpy(d_mPolyForwardProj.foSpOffsetU, h_mPolyForwardProj.foSpOffsetU, sizeFoSpOffset, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.foSpOffsetU hipMemcpy failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMemcpy(d_mPolyForwardProj.foSpOffsetV, h_mPolyForwardProj.foSpOffsetV, sizeFoSpOffset, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.foSpOffsetV hipMemcpy failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	// Kernel parameters
	dim3 blockSizeTrans(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeTrans((mCTScanSystemInfo.intNum - 1) / blockSizeTrans.x + 1, (mCTScanParas.dNumU - 1) / blockSizeTrans.y + 1, (mCTScanParas.dNumV - 1) / blockSizeTrans.z + 1);

	dim3 blockSizeProj(BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1);

	float angle = 0.0f;
	for (size_t i = 0; i < mCTScanParas.projNum; i++)
	{

		//angle = mCTScanSystemInfo.rotatedDirection * i * mCTScanSystemInfo.thetaStep / 180 * PI;   // ���㵱ǰ��ת�ǣ���ת��Ϊ������

		transformFocalSpotKernel << <gridSizeTrans, blockSizeTrans >> > (d_mPolyForwardProj, d_mCoordinate, texObj, mCTScanSystemInfo, mCTScanParas, i);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "transformFocalSpotKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transformFocalSpotKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
		//hipMemcpy(tempIntX, d_mPolyForwardProj->phantom, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);
		//FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(tempIntX, 1, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), fp);
		//fclose(fp);

		computeSinMatIndensityProjKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas, i);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "forwardProjKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardProjKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//hipMemcpy(h_mPolyForwardProj->I, d_mPolyForwardProj->I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
		//// ���
		////FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(h_mPolyForwardProj->I, 1, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), fp);
		//fclose(fp);
	}

	// ����ͶӰ����ƫ�ƣ���������ƫ�ƣ��൱�������̽����ͬʱ��ͬһ����ƫ��
	createTexture3D(texObj, d_mPolyForwardProj.proj, mCTScanParas.dNumU, mCTScanParas.dNumV, mCTScanParas.projNum);

	dim3 blockSizePOffsetMatch(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizePOffsetMatch((mCTScanParas.dNumU - 1) / blockSizePOffsetMatch.x + 1, (mCTScanParas.dNumV - 1) / blockSizePOffsetMatch.y + 1, (mCTScanParas.projNum - 1) / blockSizePOffsetMatch.z + 1);

	projOffsetMatchKernel << <gridSizePOffsetMatch, blockSizePOffsetMatch >> > (d_mPolyForwardProj, texObj, mCTScanParas);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "projOffsetMatchKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching projOffsetMatchKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>�ܶ�ͼ��ͶӰ��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);



	// Allocate GPU buffers and host
	hipMalloc(&d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));
	hipMalloc(&d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));

	// դ����
	//hipMalloc(&d_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float));

	// ̽������Ӧ����
	hipMalloc(&d_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float));

}

void initDeviceSinMatNoResponseNoGrid(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, PolyForwardProj& h_mPolyForwardProj)
{
	hipError_t cudaStatus;

	// Select Gpu
	cudaStatus = hipSetDevice(GPUINDEX);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed? \n Error: %s\n", hipGetErrorString(cudaStatus));
	}

	// ��ʼ��device
	hipDeviceSynchronize();    // ȷ����CUDA��������
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Error: %s\n", hipGetErrorString(cudaStatus));
	}
	hipDeviceReset();  // ������


	// Allocate GPU buffers
	size_t sizeIntPoint = mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float);

	// ���ֵ�����
	cudaStatus = hipMalloc(&d_mCoordinate.imgIntX, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->x hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntY, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->y hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	cudaStatus = hipMalloc(&d_mCoordinate.imgIntZ, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable imageCoordinate->z hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}


	/*hipMalloc(&d_mCoordinate->detU, mCTScanParas.dNumU * sizeof(float));
	hipMalloc(&d_mCoordinate->detV, mCTScanParas.dNumV * sizeof(float));*/

	//hipMallocManaged();

	dim3 blockSizeIPC(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeIPC((mCTScanSystemInfo.intNum - 1) / blockSizeIPC.x + 10, (mCTScanParas.dNumU - 1) / blockSizeIPC.y + 10, (mCTScanParas.dNumV - 1) / blockSizeIPC.z + 10);

	computeIntPointCoordinatesKernel << <gridSizeIPC, blockSizeIPC >> > (d_mCoordinate, mCTScanSystemInfo, mCTScanParas);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "computeIntPointCoordinatesKernel launch failed: %s! \n\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching computeIntPointCoordinatesKernel!\n Error: %s!\n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���
	/*float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntX, d_mCoordinate->imgIntX, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntY = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntY, d_mCoordinate->imgIntY, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);

	float* tempIntZ = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
	hipMemcpy(tempIntZ, d_mCoordinate->imgIntZ, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);*/


	// ��ʼ������
	volumeSize = make_hipExtent(mCTScanParas.pNumX, mCTScanParas.pNumY, mCTScanParas.pNumZ);
	initTexture3D(texObj, d_cuArray3D, h_mPolyForwardProj.phantom, volumeSize);


	// Allocate GPU buffers for temporary output variables
	cudaStatus = hipMalloc(&d_mPolyForwardProj.phantom, sizeIntPoint);   // �洢���ֵ㴦����ֵ��ת�������
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.phantom hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// ���� �ܶ�ͼ���Proj
	// �ܶ�ͼ���Proj
	hipMalloc(&d_mPolyForwardProj.proj, sizeIntPoint);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "Variable d_mPolyForwardProj.proj hipMalloc failed! \n Error Code: %d --- %s! \n\n", cudaStatus, hipGetErrorString(cudaStatus));
	}

	// GPU��ʱ
	hipEvent_t g_start, g_stop;
	hipEventCreate(&g_start);
	hipEventCreate(&g_stop);
	hipEventRecord(g_start, 0);

	// Kernel parameters
	dim3 blockSizeTrans(BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeTrans((mCTScanSystemInfo.intNum - 1) / blockSizeTrans.x + 1, (mCTScanParas.dNumU - 1) / blockSizeTrans.y + 1, (mCTScanParas.dNumV - 1) / blockSizeTrans.z + 1);

	dim3 blockSizeProj(BLOCKSIZEY, BLOCKSIZEZ);
	dim3 gridSizeProj((mCTScanParas.dNumU - 1) / blockSizeProj.x + 1, (mCTScanParas.dNumV - 1) / blockSizeProj.y + 1);

	float angle = 0.0f;
	for (size_t i = 0; i < mCTScanParas.projNum; i++)
	{

		angle = mCTScanSystemInfo.rotatedDirection * i * mCTScanSystemInfo.thetaStep / 180 * PI;   // ���㵱ǰ��ת�ǣ���ת��Ϊ������

		transformKernel << <gridSizeTrans, blockSizeTrans >> > (d_mPolyForwardProj, d_mCoordinate, texObj, mCTScanSystemInfo, mCTScanParas, angle);

		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "transformKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching transformKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//float* tempIntX = new float[mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV];
		//hipMemcpy(tempIntX, d_mPolyForwardProj->phantom, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), hipMemcpyDeviceToHost);
		//FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(tempIntX, 1, mCTScanSystemInfo.intNum * mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float), fp);
		//fclose(fp);

		computeSinMatIndensityProjKernel << <gridSizeProj, blockSizeProj >> > (d_mPolyForwardProj, mCTScanSystemInfo, mCTScanParas, i);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "forwardProjKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forwardProjKernel!\nError: %s!\n", cudaStatus, hipGetErrorString(cudaStatus));
		}

		//// ���Դ���
		//hipMemcpy(h_mPolyForwardProj->I, d_mPolyForwardProj->I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), hipMemcpyDeviceToHost);
		//// ���
		////FILE* fp;
		//fp = fopen("test.raw", "wb");
		//fwrite(h_mPolyForwardProj->I, 1, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float), fp);
		//fclose(fp);
	}

	// ��ʱ
	hipEventRecord(g_stop, 0);
	hipEventSynchronize(g_stop);
	float elapsedTime = 0;
	hipEventElapsedTime(&elapsedTime, g_start, g_stop);
	std::cout << "==>>�ܶ�ͼ��ͶӰ��ʱ(GPU)��" << elapsedTime / 1000.0f << " s" << std::endl;
	hipEventDestroy(g_start);
	hipEventDestroy(g_stop);



	// Allocate GPU buffers and host
	hipMalloc(&d_mPolyForwardProj.I, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));
	hipMalloc(&d_mPolyForwardProj.IAbsorb, mCTScanParas.dNumU * mCTScanParas.dNumV * mCTScanParas.projNum * sizeof(float));

	// դ����
	//hipMalloc(&d_mPolyForwardProj.grid, mCTScanParas.dNumU * sizeof(float));

}

void mallocDetResponse(PolyForwardProj& d_mPolyForwardProj, CTScanParas mCTScanParas)
{
	// ̽������Ӧ����
	hipMalloc(&d_mPolyForwardProj.detResponse, mCTScanParas.dNumU * mCTScanParas.dNumV * sizeof(float));
}



void freeDeviceMemory(PolyForwardProj& d_mPolyForwardProj, Coordinate& d_mCoordinate)
{
	CUDAFREE(d_mCoordinate.imgIntX);
	CUDAFREE(d_mCoordinate.imgIntY);
	CUDAFREE(d_mCoordinate.imgIntZ);

	CUDAFREE(d_mPolyForwardProj.I);
	CUDAFREE(d_mPolyForwardProj.I0);
	CUDAFREE(d_mPolyForwardProj.IAbsorb);
	CUDAFREE(d_mPolyForwardProj.proj);
	CUDAFREE(d_mPolyForwardProj.phantom);
	CUDAFREE(d_mPolyForwardProj.phantomMassAtten);
	CUDAFREE(d_mPolyForwardProj.spectrumNormal);
	CUDAFREE(d_mPolyForwardProj.detResponse);
	CUDAFREE(d_mPolyForwardProj.grid);
	CUDAFREE(d_mPolyForwardProj.gridLinearAtten);
	CUDAFREE(d_mPolyForwardProj.scintillatorLineAtten);
	CUDAFREE(d_mPolyForwardProj.scintillatorPerThickness);
	CUDAFREE(d_mPolyForwardProj.foSpOffsetU);
	CUDAFREE(d_mPolyForwardProj.foSpOffsetV);

	hipDestroyTextureObject(texObj);
	hipFreeArray(d_cuArray3D);
}

// --------------------------------------------------------------------------------------------------

__global__ void computeIntPointCoordinatesKernel(Coordinate d_Coordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas)
{
	// ��������
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x < mCTScanSystemInfo.intNum && y < mCTScanParas.dNumU && z < mCTScanParas.dNumV)
	{

		// ̽������Ԫ����
		// ����ģ����ϵ��������任
		float detU = mCTScanSystemInfo.dHalfLU - mCTScanParas.dSize / 2 - y * mCTScanParas.dSize;
		float detV = mCTScanSystemInfo.dHalfLV - mCTScanParas.dSize / 2 - z * mCTScanParas.dSize;



		// ֻ�Ǵ˴����õ���ʱ��������˲���Ҫ�洢
		/*d_Coordinate.detU[y] = mCTScanSystemInfo.dHalfLU - mCTScanParas.dSize / 2 - y * mCTScanParas.dSize;
		d_Coordinate.detV[z] = mCTScanSystemInfo.dHalfLV - mCTScanParas.dSize / 2 - z * mCTScanParas.dSize;*/

		// Compute trigonometric value of Gamma angle and Beta angle
		// Gamma represents the angle between the ray and the xoy plane
		// Beta represents the angle between the ray and the xoz plane
		float sinGamma = detV / sqrtf(powf(mCTScanParas.sdd, 2) + powf(detU, 2) + powf(detV, 2));
		float cosGamma = sqrtf(powf(mCTScanParas.sdd, 2) + powf(detU, 2)) / sqrtf(powf(mCTScanParas.sdd, 2) + powf(detU, 2) + powf(detV, 2));

		float sinBeta = detU / sqrtf(powf(mCTScanParas.sdd, 2) + powf(detU, 2) + powf(detV, 2));
		float cosBeta = sqrtf(powf(mCTScanParas.sdd, 2) + powf(detV, 2)) / sqrtf(powf(mCTScanParas.sdd, 2) + powf(detU, 2) + powf(detV, 2));


		/*float sinGamma = d_Coordinate.detV[z] / sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate.detU[y], 2) + powf(d_Coordinate.detV[z], 2));
		float cosGamma = sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate.detU[y], 2)) / sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate.detU[y], 2) + powf(d_Coordinate.detV[z], 2));

		float sinBeta = d_Coordinate.detU[y] / sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate.detU[y], 2) + powf(d_Coordinate.detV[z], 2));
		float cosBeta = sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate.detV[z], 2)) / sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate.detU[y], 2) + powf(d_Coordinate.detV[z], 2));*/


		/*float sinBeta = detY[y] / sqrtf(powf(sdd, 2) + powf(detY[y], 2) + powf(detZ[z], 2));
		float cosBeta = sqrtf(powf(sdd, 2) + powf(detZ[z], 2)) / sqrtf(powf(sdd, 2) + powf(detY[y], 2) + powf(detZ[z], 2));*/


		// Compute integration point coordinates
		size_t index = z * mCTScanSystemInfo.intNum * mCTScanParas.dNumU + y * mCTScanSystemInfo.intNum + x;

		d_Coordinate.imgIntX[index] = cosGamma * cosBeta * (mCTScanParas.sod - mCTScanSystemInfo.FOVR + mCTScanSystemInfo.dx / 2 + x * mCTScanSystemInfo.dx) - mCTScanParas.sod;  // ��
		d_Coordinate.imgIntY[index] = cosGamma * sinBeta * (mCTScanParas.sod - mCTScanSystemInfo.FOVR + mCTScanSystemInfo.dx / 2 + x * mCTScanSystemInfo.dx);  // ��
		d_Coordinate.imgIntZ[index] = sinGamma * (mCTScanParas.sod - mCTScanSystemInfo.FOVR + mCTScanSystemInfo.dx / 2 + x * mCTScanSystemInfo.dx);  // ҳ
	}
}


//__global__ void computeIntPointCoordinatesKernel(Coordinate* d_Coordinate, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas)
//{
//	// ��������
//	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
//	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;
//
//	if (x < mCTScanSystemInfo.intNum && y < mCTScanParas.dNumU && z < mCTScanParas.dNumV)
//	{
//
//		// ̽������Ԫ����
//		// ����ģ����ϵ��������任
//		/*float detU = mCTScanSystemInfo.dHalfLU / 2 - y * mCTScanParas.dSize;
//		float detV = mCTScanSystemInfo.dHalfLV / 2 - z * mCTScanParas.dSize;*/
//
//		d_Coordinate->detU[y] = mCTScanSystemInfo.dHalfLU / 2 - y * mCTScanParas.dSize;
//		d_Coordinate->detV[z] = mCTScanSystemInfo.dHalfLV / 2 - z * mCTScanParas.dSize;
//
//		// Compute trigonometric value of Gamma angle and Beta angle
//		// Gamma represents the angle between the ray and the xoy plane
//		// Beta represents the angle between the ray and the xoz plane
//		/*float sinGamma = detV / sqrtf(powf(mCTScanParas.sdd, 2) + powf(detU, 2) + powf(detV, 2));
//		float cosGamma = sqrtf(powf(mCTScanParas.sdd, 2) + powf(detU, 2)) / sqrtf(powf(mCTScanParas.sdd, 2) + powf(detU, 2) + powf(detV, 2));
//
//		float sinBeta = detU / sqrtf(powf(mCTScanParas.sdd, 2) + powf(detU, 2) + powf(detV, 2));
//		float cosBeta = sqrtf(powf(mCTScanParas.sdd, 2) + powf(detV, 2)) / sqrtf(powf(mCTScanParas.sdd, 2) + powf(detU, 2) + powf(detV, 2));*/
//
//
//		float sinGamma = d_Coordinate->detU[y] / sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate->detU[y], 2) + powf(d_Coordinate->detV[z], 2));
//		float cosGamma = sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate->detU[y], 2)) / sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate->detU[y], 2) + powf(d_Coordinate->detV[z], 2));
//
//		float sinBeta = d_Coordinate->detU[y] / sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate->detU[y], 2) + powf(d_Coordinate->detV[z], 2));
//		float cosBeta = sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate->detV[z], 2)) / sqrtf(powf(mCTScanParas.sdd, 2) + powf(d_Coordinate->detU[y], 2) + powf(d_Coordinate->detV[z], 2));
//
//
//		/*float sinBeta = detY[y] / sqrtf(powf(sdd, 2) + powf(detY[y], 2) + powf(detZ[z], 2));
//		float cosBeta = sqrtf(powf(sdd, 2) + powf(detZ[z], 2)) / sqrtf(powf(sdd, 2) + powf(detY[y], 2) + powf(detZ[z], 2));*/
//
//
//		// Compute integration point coordinates
//		size_t index = z * mCTScanParas.dNumU * mCTScanParas.dNumV + y * mCTScanParas.dNumU + x;
//
//		d_Coordinate->imgIntX[index] = cosGamma * cosBeta * (mCTScanParas.sod - mCTScanSystemInfo.FOVR + mCTScanSystemInfo.dx / 2 + x * mCTScanSystemInfo.dx) - mCTScanParas.sod;  // ��
//		d_Coordinate->imgIntY[index] = cosGamma * sinBeta * (mCTScanParas.sod - mCTScanSystemInfo.FOVR + mCTScanSystemInfo.dx / 2 + x * mCTScanSystemInfo.dx);  // ��
//		d_Coordinate->imgIntZ[index] = sinGamma * (mCTScanParas.sod - mCTScanSystemInfo.FOVR + mCTScanSystemInfo.dx / 2 + x * mCTScanSystemInfo.dx);  // ҳ
//	}
//}


__global__ void transformKernel(PolyForwardProj d_mPolyForwardProj, Coordinate d_Coordinate, hipTextureObject_t texObj, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, float theta)
{
	// ������������
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x < mCTScanSystemInfo.intNum && y < mCTScanParas.dNumU && z < mCTScanParas.dNumV)
	{
		// �ǹ�һ��
		size_t index = z * mCTScanSystemInfo.intNum * mCTScanParas.dNumU + y * mCTScanSystemInfo.intNum + x;


		float tu = d_Coordinate.imgIntX[index] * cosf(theta) - d_Coordinate.imgIntY[index] * sinf(theta) + mCTScanSystemInfo.pHalfX;// mCTScanSystemInfo.FOVR;
		float tv = -(d_Coordinate.imgIntX[index] * sinf(theta) + d_Coordinate.imgIntY[index] * cosf(theta)) + mCTScanSystemInfo.pHalfY;//mCTScanSystemInfo.FOVR;     // ��ͶӰģ���У�ʹ������ϵ���˴��ı�y�᷽�򣬱������ʹ�õ�����ϵ
		float tw = -d_Coordinate.imgIntZ[index] + mCTScanSystemInfo.pHalfZ; //mCTScanSystemInfo.FOVH;   // �ı�Z�᷽�򣬷�������Ϊ������Ԥ�ڽ�ģһ��

		// �������ж�ȡ��д��ȫ�ִ洢
		d_mPolyForwardProj.phantom[index] = tex3D<float>(texObj, tu / mCTScanSystemInfo.pSizeX + 0.5, tv / mCTScanSystemInfo.pSizeY + 0.5, tw / mCTScanSystemInfo.pSizeZ + 0.5); //  �Ƿ���Ҫ��0.5����������   �������0.5Ҫ��Ϊ���ش�С
		// ����Y���ּ���������������ת�任ʱӦ��ȥ����ֵ����ͳһ�����ڲ�ֵ����ʱ��Ҫ���������ֵ�����Ե��������������ʡ�ԣ�X����Ϊ�˷���Ҳ��Y����ͬ��ʡ��
		// ���
	}
}

// ������ƫ��
__global__ void transformFocalSpotKernel(PolyForwardProj d_mPolyForwardProj, Coordinate d_Coordinate, hipTextureObject_t texObj, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, int angleIndex)
{
	// ������������
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	float theta = mCTScanSystemInfo.rotatedDirection * angleIndex * mCTScanSystemInfo.thetaStep / 180.0f * PI;

	if (x < mCTScanSystemInfo.intNum && y < mCTScanParas.dNumU && z < mCTScanParas.dNumV)
	{
		// �ǹ�һ��
		size_t index = z * mCTScanSystemInfo.intNum * mCTScanParas.dNumU + y * mCTScanSystemInfo.intNum + x;


		float tx = d_Coordinate.imgIntX[index] * cosf(theta) - d_Coordinate.imgIntY[index] * sinf(theta) + mCTScanSystemInfo.pHalfX;// mCTScanSystemInfo.FOVR;
		float ty = -(d_Coordinate.imgIntX[index] * sinf(theta) + d_Coordinate.imgIntY[index] * cosf(theta)) + mCTScanSystemInfo.pHalfY + d_mPolyForwardProj.foSpOffsetU[angleIndex];//mCTScanSystemInfo.FOVR;     // ��ͶӰģ���У�ʹ������ϵ���˴��ı�y�᷽�򣬱������ʹ�õ�����ϵ
		float tz = -d_Coordinate.imgIntZ[index] + mCTScanSystemInfo.pHalfZ + d_mPolyForwardProj.foSpOffsetV[angleIndex]; //mCTScanSystemInfo.FOVH;   // �ı�Z�᷽�򣬷�������Ϊ������Ԥ�ڽ�ģһ��
		// foSpOffset ����ƫ����


		// �������ж�ȡ��д��ȫ�ִ洢
		d_mPolyForwardProj.phantom[index] = tex3D<float>(texObj, tx / mCTScanSystemInfo.pSizeX + 0.5, ty / mCTScanSystemInfo.pSizeY + 0.5, tz / mCTScanSystemInfo.pSizeZ + 0.5);
		// ����Y���ּ���������������ת�任ʱӦ��ȥ����ֵ����ͳһ�����ڲ�ֵ����ʱ��Ҫ���������ֵ�����Ե��������������ʡ�ԣ�X����Ϊ�˷���Ҳ��Y����ͬ��ʡ��
		// ���
	}

}

__global__ void projOffsetMatchKernel(PolyForwardProj d_mPolyForwardProj, hipTextureObject_t texObj, CTScanParas mCTScanParas)
{
	// ������������
	unsigned int u = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int v = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int pn = blockIdx.z * blockDim.z + threadIdx.z;

	if (u < mCTScanParas.dNumU && v < mCTScanParas.dNumV && pn < mCTScanParas.projNum)
	{
		// �ǹ�һ��

		size_t index = pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u;

		// foSpOffset ����ƫ����

		// �������ж�ȡ��д��ȫ�ִ洢
		d_mPolyForwardProj.phantom[index]
			= tex3D<float>(texObj
				, u + d_mPolyForwardProj.foSpOffsetU[pn] / mCTScanParas.dSize + 0.5
				, v + d_mPolyForwardProj.foSpOffsetV[pn] / mCTScanParas.dSize + 0.5
				, pn + 0.5);
	}
}

__global__ void forwardProjGridKernel(PolyForwardProj d_mPolyForwardProj, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, size_t num)
{
	// ��X���������
	size_t y = blockIdx.x * blockDim.x + threadIdx.x;
	size_t z = blockIdx.y * blockDim.y + threadIdx.y;

	if (y < mCTScanParas.dNumU && z < mCTScanParas.dNumV)
	{
		float temp = 0.0f;
		for (size_t i = 0; i < mCTScanSystemInfo.intNum; i++)
		{
			temp += d_mPolyForwardProj.phantom[z * mCTScanSystemInfo.intNum * mCTScanParas.dNumU + y * mCTScanSystemInfo.intNum + i];
		}

		d_mPolyForwardProj.I[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * expf(-temp * mCTScanSystemInfo.dx) * d_mPolyForwardProj.grid[y] * d_mPolyForwardProj.detResponse[z * mCTScanParas.dNumU + y];  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		d_mPolyForwardProj.IAbsorb[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * (1 - expf(-temp * mCTScanSystemInfo.dx)) * d_mPolyForwardProj.grid[y] * d_mPolyForwardProj.detResponse[z * mCTScanParas.dNumU + y];

		// 
		//temp = mCTScanParas.I0Val * expf(-temp * mCTScanSystemInfo.dx);
		//d_mPolyForwardProj.I[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanSystemInfo.spectrumVal * temp * d_mPolyForwardProj.grid[y];  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		//d_mPolyForwardProj.IAbsorb[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanSystemInfo.spectrumVal * (mCTScanParas.I0Val - temp) * d_mPolyForwardProj.grid[y];

	}
}

__global__ void computeSinMatIndensityProjKernel(PolyForwardProj d_mPolyForwardProj, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, size_t num)
{
	// ��X���������
	size_t u = blockIdx.x * blockDim.x + threadIdx.x;
	size_t v = blockIdx.y * blockDim.y + threadIdx.y;

	if (u < mCTScanParas.dNumU && v < mCTScanParas.dNumV)
	{
		float temp = 0.0f;
		for (size_t i = 0; i < mCTScanSystemInfo.intNum; i++)
		{
			temp += d_mPolyForwardProj.phantom[v * mCTScanSystemInfo.intNum * mCTScanParas.dNumU + u * mCTScanSystemInfo.intNum + i];
		}
		d_mPolyForwardProj.proj[num * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u] = temp * mCTScanSystemInfo.dx;    // �ܶȺ� �� ��������
	}
}

__global__ void forwardSinMatProjGridKernel(PolyForwardProj d_mPolyForwardProj, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, size_t num)
{
	// ��X���������
	size_t y = blockIdx.x * blockDim.x + threadIdx.x;
	size_t z = blockIdx.y * blockDim.y + threadIdx.y;

	if (y < mCTScanParas.dNumU && z < mCTScanParas.dNumV)
	{
		float temp = 0.0f;
		for (size_t i = 0; i < mCTScanSystemInfo.intNum; i++)
		{
			temp += d_mPolyForwardProj.phantom[z * mCTScanSystemInfo.intNum * mCTScanParas.dNumU + y * mCTScanSystemInfo.intNum + i];
		}

		d_mPolyForwardProj.I[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * expf(-mCTScanSystemInfo.phantomMAtten * temp * mCTScanSystemInfo.dx) * d_mPolyForwardProj.grid[y] * d_mPolyForwardProj.detResponse[z * mCTScanParas.dNumU + y];  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		d_mPolyForwardProj.IAbsorb[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * (1 - expf(-mCTScanSystemInfo.phantomMAtten * temp * mCTScanSystemInfo.dx)) * d_mPolyForwardProj.grid[y] * d_mPolyForwardProj.detResponse[z * mCTScanParas.dNumU + y];

		// 
		//temp = mCTScanParas.I0Val * expf(-temp * mCTScanSystemInfo.dx);
		//d_mPolyForwardProj.I[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanSystemInfo.spectrumVal * temp * d_mPolyForwardProj.grid[y];  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		//d_mPolyForwardProj.IAbsorb[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanSystemInfo.spectrumVal * (mCTScanParas.I0Val - temp) * d_mPolyForwardProj.grid[y];
	}
}

__global__ void forwardSinMatProjGridKernel(PolyForwardProj d_mPolyForwardProj, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas)
{
	// ��X���������
	size_t u = blockIdx.x * blockDim.x + threadIdx.x;
	size_t v = blockIdx.y * blockDim.y + threadIdx.y;
	size_t pn = blockIdx.z * blockDim.z + threadIdx.z;

	if (u < mCTScanParas.dNumU && v < mCTScanParas.dNumV && pn < mCTScanParas.projNum)
	{
		d_mPolyForwardProj.I[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * expf(-mCTScanSystemInfo.phantomMAtten * d_mPolyForwardProj.proj[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u]) * d_mPolyForwardProj.grid[u] * d_mPolyForwardProj.detResponse[v * mCTScanParas.dNumU + u];  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		d_mPolyForwardProj.IAbsorb[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * (1 - expf(-mCTScanSystemInfo.phantomMAtten * d_mPolyForwardProj.proj[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u])) * d_mPolyForwardProj.grid[u] * d_mPolyForwardProj.detResponse[v * mCTScanParas.dNumU + u];

		// 
		//temp = mCTScanParas.I0Val * expf(-temp * mCTScanSystemInfo.dx);
		//d_mPolyForwardProj.I[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanSystemInfo.spectrumVal * temp * d_mPolyForwardProj.grid[y];  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		//d_mPolyForwardProj.IAbsorb[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanSystemInfo.spectrumVal * (mCTScanParas.I0Val - temp) * d_mPolyForwardProj.grid[y];
	}
}

__global__ void forwardSinMatNoResponseProjGridKernel(PolyForwardProj d_mPolyForwardProj, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas)
{
	// ��X���������
	size_t u = blockIdx.x * blockDim.x + threadIdx.x;
	size_t v = blockIdx.y * blockDim.y + threadIdx.y;
	size_t pn = blockIdx.z * blockDim.z + threadIdx.z;

	if (u < mCTScanParas.dNumU && v < mCTScanParas.dNumV && pn < mCTScanParas.projNum)
	{
		d_mPolyForwardProj.I[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * expf(-mCTScanSystemInfo.phantomMAtten * d_mPolyForwardProj.proj[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u]) * d_mPolyForwardProj.grid[u];  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		d_mPolyForwardProj.IAbsorb[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * (1 - expf(-mCTScanSystemInfo.phantomMAtten * d_mPolyForwardProj.proj[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u])) * d_mPolyForwardProj.grid[u];
	}
}

__global__ void forwardProjNoGridKernel(PolyForwardProj d_mPolyForwardProj, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, size_t num)
{
	// ��X���������
	size_t y = blockIdx.x * blockDim.x + threadIdx.x;
	size_t z = blockIdx.y * blockDim.y + threadIdx.y;

	if (y < mCTScanParas.dNumU && z < mCTScanParas.dNumV)
	{
		float temp = 0.0f;
		for (size_t i = 0; i < mCTScanSystemInfo.intNum; i++)
		{
			temp += d_mPolyForwardProj.phantom[z * mCTScanSystemInfo.intNum * mCTScanParas.dNumU + y * mCTScanSystemInfo.intNum + i];
		}

		d_mPolyForwardProj.I[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y]
			= mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal
			* expf(-temp * mCTScanSystemInfo.dx)
			* d_mPolyForwardProj.detResponse[z * mCTScanParas.dNumU + y];  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		d_mPolyForwardProj.IAbsorb[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y]
			= mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal
			* (1 - expf(-temp * mCTScanSystemInfo.dx))
			* d_mPolyForwardProj.detResponse[z * mCTScanParas.dNumU + y];

		// 
		//temp = mCTScanParas.I0Val * expf(-temp * mCTScanSystemInfo.dx);
		//d_mPolyForwardProj.I[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanSystemInfo.spectrumVal * temp * d_mPolyForwardProj.grid[y];  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		//d_mPolyForwardProj.IAbsorb[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanSystemInfo.spectrumVal * (mCTScanParas.I0Val - temp) * d_mPolyForwardProj.grid[y];

	}
}

__global__ void forwardSinMatProjNoGridKernel(PolyForwardProj d_mPolyForwardProj, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas, size_t num)
{
	// ��X���������
	size_t y = blockIdx.x * blockDim.x + threadIdx.x;
	size_t z = blockIdx.y * blockDim.y + threadIdx.y;

	if (y < mCTScanParas.dNumU && z < mCTScanParas.dNumV)
	{
		float temp = 0.0f;
		for (size_t i = 0; i < mCTScanSystemInfo.intNum; i++)
		{
			temp += d_mPolyForwardProj.phantom[z * mCTScanSystemInfo.intNum * mCTScanParas.dNumU + y * mCTScanSystemInfo.intNum + i];
		}

		d_mPolyForwardProj.I[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y]
			= mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal
			* expf(-mCTScanSystemInfo.phantomMAtten * temp * mCTScanSystemInfo.dx)
			* d_mPolyForwardProj.detResponse[z * mCTScanParas.dNumU + y];         // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		d_mPolyForwardProj.IAbsorb[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y]
			= mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal
			* (1 - expf(-mCTScanSystemInfo.phantomMAtten * temp * mCTScanSystemInfo.dx))
			* d_mPolyForwardProj.detResponse[z * mCTScanParas.dNumU + y];

		// 
		//temp = mCTScanParas.I0Val * expf(-temp * mCTScanSystemInfo.dx);
		//d_mPolyForwardProj.I[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanSystemInfo.spectrumVal * temp * d_mPolyForwardProj.grid[y];  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		//d_mPolyForwardProj.IAbsorb[num * mCTScanParas.dNumU * mCTScanParas.dNumV + z * mCTScanParas.dNumU + y] = mCTScanSystemInfo.spectrumVal * (mCTScanParas.I0Val - temp) * d_mPolyForwardProj.grid[y];

	}
}

__global__ void forwardSinMatProjNoGridKernel(PolyForwardProj d_mPolyForwardProj, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas)
{
	// ��X���������
	size_t u = blockIdx.x * blockDim.x + threadIdx.x;
	size_t v = blockIdx.y * blockDim.y + threadIdx.y;
	size_t pn = blockIdx.z * blockDim.z + threadIdx.z;

	if (u < mCTScanParas.dNumU && v < mCTScanParas.dNumV && pn < mCTScanParas.projNum)
	{
		d_mPolyForwardProj.I[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * expf(-mCTScanSystemInfo.phantomMAtten * d_mPolyForwardProj.proj[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u]) * d_mPolyForwardProj.detResponse[v * mCTScanParas.dNumU + u];  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		d_mPolyForwardProj.IAbsorb[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * (1 - expf(-mCTScanSystemInfo.phantomMAtten * d_mPolyForwardProj.proj[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u])) * d_mPolyForwardProj.detResponse[v * mCTScanParas.dNumU + u];

	}

}

__global__ void forwardSinMatNoResponseProjNoGridKernel(PolyForwardProj d_mPolyForwardProj, CTScanSystemInfo mCTScanSystemInfo, CTScanParas mCTScanParas)
{
	// ��X���������
	size_t u = blockIdx.x * blockDim.x + threadIdx.x;
	size_t v = blockIdx.y * blockDim.y + threadIdx.y;
	size_t pn = blockIdx.z * blockDim.z + threadIdx.z;

	if (u < mCTScanParas.dNumU && v < mCTScanParas.dNumV && pn < mCTScanParas.projNum)
	{
		d_mPolyForwardProj.I[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * expf(-mCTScanSystemInfo.phantomMAtten * d_mPolyForwardProj.proj[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u]);  // ��X�������, ÿһ���Ƕ���Ϊһ��, Y����Ϊ�У� Z����Ϊ��
		d_mPolyForwardProj.IAbsorb[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u] = mCTScanParas.I0Val * mCTScanSystemInfo.spectrumVal * (1 - expf(-mCTScanSystemInfo.phantomMAtten * d_mPolyForwardProj.proj[pn * mCTScanParas.dNumU * mCTScanParas.dNumV + v * mCTScanParas.dNumU + u]));
	}
}



// ��ʼ����ά����, �������Դ����, ��������ʱֻ�����cuArray, ���ݴ����� Host to Device.
// texObj -- �������, d_cuArray3D -- Device�д洢���ݵ�ָ��, data -- Դ����, volumeSize -- �����С.
void initTexture3D(hipTextureObject_t& texObj, hipArray_t& d_cuArray3D, float* h_data, hipExtent volumeSize)
{
	//hipExtent volumeSize = make_hipExtent(mCTScanParas.pNumX, mCTScanParas.pNumY, mCTScanParas.pNumZ);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	//hipArray* d_cuArray3D;
	hipMalloc3DArray(&d_cuArray3D, &channelDesc, volumeSize);

	// �������ݵ�CUDA array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr((void*)h_data, volumeSize.width * sizeof(float), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_cuArray3D;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// ������Դ������
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(hipResourceDesc));

	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = d_cuArray3D;

	// ��������������
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(hipTextureDesc));
	texDesc.addressMode[0] = hipAddressModeBorder;
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.addressMode[2] = hipAddressModeBorder;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	// �����������
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
}

// ��������
void updateTex(hipArray_t& d_cuArray3D, float* h_data, hipExtent volumeSize)
{
	// �������ݵ�CUDA array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr((void*)h_data, volumeSize.width * sizeof(float), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_cuArray3D;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
}

void createTexture3D(hipTextureObject_t& texObj, float* h_data, size_t width, size_t height, size_t depth)
{
	hipExtent volumeSize = make_hipExtent(width, height, depth);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray* d_cuArray3D;
	hipMalloc3DArray(&d_cuArray3D, &channelDesc, volumeSize);

	// �������ݵ�CUDA array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr((void*)h_data, volumeSize.width * sizeof(float), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_cuArray3D;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	// ������Դ������
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(hipResourceDesc));

	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = d_cuArray3D;

	// ��������������
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(hipTextureDesc));
	texDesc.addressMode[0] = hipAddressModeBorder;
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.addressMode[2] = hipAddressModeBorder;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	// �����������
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
}




